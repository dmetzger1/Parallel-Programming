#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16
#define BLOCK_SIZE 256

__global__ void matrix_unrolling_kernel(const float *input, float *output,
                                        const int Batch, const int Channel,
                                        const int Height, const int Width,
                                        const int K) {
    /*
    Modify this function to implement the input matrix unrolling kernel.

    Function paramter definitions:
    input - input
    output - output
    Batch - batch_size (number of images in x)
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    (void)Width_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)

    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]

    // TODO: Insert your input matrix unrolling kernel code here
    // each thread computes K* K number of elements
    size_t cur_channel, s, h_out_idx, w_out_idx, w_base, w_unroll_idx;
    size_t t = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int batch_num = blockIdx.y;
    size_t W_unroll_per_matrix = Height_out * Width_out; // the width of one unrolled matrix as a whole for one batch

    if(t < Channel * W_unroll_per_matrix){
        cur_channel = t / W_unroll_per_matrix;
        s = t%W_unroll_per_matrix;
        h_out_idx = s/Width_out;
        w_out_idx = s%Width_out;
        w_unroll_idx = h_out_idx * Width_out + w_out_idx;
        w_base = cur_channel * K * K;
        size_t h_unroll_idx = 0;
        #pragma unroll
        for(int p = 0; p < K; p++){     // for each element of the KxK filter (two loops)
            #pragma unroll
            for(int q = 0; q < K; q++){
                h_unroll_idx = w_base + p*K + q;
                // int input_val = in_4d(batch_num, cur_channel, h_out_idx+p, w_out_idx+q); // get input pixel value
                // int output_index = W_unroll_per_matrix * Batch * h_unroll_idx + batch_num * W_unroll_per_matrix + w_unroll_idx;
                output[(size_t)(W_unroll_per_matrix * Batch * h_unroll_idx + batch_num * W_unroll_per_matrix + w_unroll_idx)] = in_4d(batch_num, cur_channel, h_out_idx+p, w_out_idx+q);    // copy input pixels, how do i do this?
            }
        }

    }

    #undef in_4d
}

// Tiled matrix multiplication kernel. Computes C = AB
// You don't need to modify this kernel.
__global__ void matrixMultiplyShared(const float *A, const float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns)
{
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    int by = blockIdx.y, bx = blockIdx.x, ty = threadIdx.y, tx = threadIdx.x;

    int row = by * TILE_WIDTH + ty, col = bx * TILE_WIDTH + tx;
    float val = 0;

    for (int tileId = 0; tileId < (numAColumns - 1) / TILE_WIDTH + 1; tileId++) {
        if (row < numARows && tileId * TILE_WIDTH + tx < numAColumns) {
            tileA[ty][tx] = A[(size_t) row * numAColumns + tileId * TILE_WIDTH + tx];
        } else {
            tileA[ty][tx] = 0;
        }
        if (col < numBColumns && tileId * TILE_WIDTH + ty < numBRows) {
            tileB[ty][tx] = B[((size_t) tileId * TILE_WIDTH + ty) * numBColumns + col];
        } else {
            tileB[ty][tx] = 0;
        }
        __syncthreads();

        if (row < numCRows && col < numCColumns) {
            #pragma unroll
            for (int i = 0; i < TILE_WIDTH; i++) {
                val += tileA[ty][i] * tileB[i][tx];
            }
        }
        __syncthreads();
    }

    if (row < numCRows && col < numCColumns) {
        C[row * numCColumns + col] = val;
    }
}

// Permutes the matmul result.
// The output feature map after matmul is of shape Map_out x Batch x Height_out x Width_out,
// and we need to permute it into Batch x Map_out x Height_out x Width_out.
// You don't need to modify this kernel.
__global__ void matrix_permute_kernel(const float *input, float *output, int Map_out,
                                      int Batch, int image_size) {
    int b = blockIdx.y;
    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    if (x < image_size) {
        #pragma unroll
        for (int m = 0; m < Map_out; m++) {
            output[b * Map_out * image_size + m * image_size + x] =
                    input[m * Batch * image_size + b * image_size + x];
        }
    }
}

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;

    int input_size = Batch * Channel * Width * Height * sizeof(float);
    int output_size = Batch * Map_out * Height_out * Width_out * sizeof(float);
    int mask_size = Map_out * Channel * K * K * sizeof(float);

    hipMalloc((void**) device_input_ptr, input_size);
    hipMalloc((void**) device_output_ptr, output_size);
    hipMalloc((void**) device_mask_ptr, mask_size);

    hipMemcpy(*device_input_ptr, host_input, input_size, hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, mask_size, hipMemcpyHostToDevice);

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    const int Height_unrolled = Channel * K * K;
    const int Width_unrolled = Batch * Height_out * Width_out;

    int num_threads = Channel * Height_out * Width_out;
    int numBlocks = ((num_threads + BLOCK_SIZE - 1)/BLOCK_SIZE);

    float *unrolled_matrix;  // Pointer to device memory for storing the unrolled matrix
    float *matmul_output;    // Pointer to device memory for storing the result of matrix multiplication
    hipMalloc((void**)&unrolled_matrix, (size_t) Batch * Channel * K * K * Height_out * Width_out * sizeof(float));
    hipMalloc((void**)&matmul_output, (Batch * Map_out * Height_out * Width_out) * sizeof(float));

    // TODO: Set the kernel dimensions and call the matrix unrolling kernel.
    dim3 blockDim1(BLOCK_SIZE, 1, 1);
    dim3 gridDim1(numBlocks, Batch, 1);

    matrix_unrolling_kernel<<<gridDim1, blockDim1>>>(device_input, unrolled_matrix, Batch, Channel, Height, Width, K);
    hipDeviceSynchronize();

    // TODO: Set the kernel dimensions and call the matmul kernel
    dim3 gridDim2(((Width_unrolled + TILE_WIDTH -1) / TILE_WIDTH), ((Height_unrolled + TILE_WIDTH - 1)/TILE_WIDTH), 1);
    dim3 blockDim2(TILE_WIDTH, TILE_WIDTH, 1);

    matrixMultiplyShared<<<gridDim2, blockDim2>>>(device_mask, unrolled_matrix, matmul_output,
                                     Map_out, Height_unrolled,
                                     Height_unrolled, Width_unrolled,
                                     Map_out, Width_unrolled);
    hipDeviceSynchronize();

    // Permute the result of matrix multiplication
    const int out_image_size = Height_out * Width_out;
    dim3 permute_kernel_grid_dim((out_image_size - 1) / BLOCK_SIZE + 1, Batch, 1);
    matrix_permute_kernel<<<permute_kernel_grid_dim, BLOCK_SIZE>>>(
        matmul_output, device_output, Map_out, Batch, out_image_size
    );

    hipFree(matmul_output);
    hipFree(unrolled_matrix);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;
    int output_size = Batch * Map_out * Height_out * Width_out * sizeof(float);

    hipMemcpy(host_output, device_output, output_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}